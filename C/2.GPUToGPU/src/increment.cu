
#include <hip/hip_runtime.h>
// CUDA kernel to increment each element of the array by 1
__global__ void increment(int *arr, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within the array bounds
    if (tid < size) {
        arr[tid] += 1;
    }
}

extern "C" void launch_increment(int* array_device, int SIZE)
{
    // Define block and grid dimensions
    int blockSize = 256;
    int gridSize = (SIZE + blockSize - 1) / blockSize;

    // Launch the kernel
    increment<<<gridSize, blockSize>>>(array_device, SIZE);
}